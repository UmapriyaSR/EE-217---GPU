#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>


#define BLOCK_SIZE 256  // You can experiment with different block sizes

typedef int           BOOL;
typedef int           INT;
typedef double        REAL;

#define FALSE         0
#define TRUE          1
#define NOT           !
#define AND           &&
#define OR            ||

#define MIN_REAL      -HUGE_VAL
#define MAX_REAL      +HUGE_VAL
#define MIN(x,y)      ((x)<(y) ? (x) : (y))
#define MAX(x,y)      ((x)>(y) ? (x) : (y))

#define LO            0.1
#define HI            0.9
#define BIAS          1

#define sqr(x)        ((x)*(x))


typedef struct {                     /* A LAYER OF A NET:                     */
        INT           Units;         /* - number of units in this layer       */
        REAL*         Output;        /* - output of ith unit                  */
        REAL*         Error;         /* - error term of ith unit              */
        REAL**        Weight;        /* - connection weights to ith unit      */
        REAL**        WeightSave;    /* - saved weights for stopped training  */
        REAL**        dWeight;       /* - last weight deltas for momentum     */
} LAYER;

typedef struct {                     /* A NET:                                */
        LAYER**       Layer;         /* - layers of this net                  */
        LAYER*        InputLayer;    /* - input layer                         */
        LAYER*        OutputLayer;   /* - output layer                        */
        REAL          Alpha;         /* - momentum factor                     */
        REAL          Eta;           /* - learning rate                       */
        REAL          Gain;          /* - gain of sigmoid function            */
        REAL          Error;         /* - total net error                     */
} NET;


/******************************************************************************
        R A N D O M S   D R A W N   F R O M   D I S T R I B U T I O N S
 ******************************************************************************/


void InitializeRandoms()
{
  srand(4711);
}


INT RandomEqualINT(INT Low, INT High)
{
  return rand() % (High-Low+1) + Low;
}      


REAL RandomEqualREAL(REAL Low, REAL High)
{
  return ((REAL) rand() / RAND_MAX) * (High-Low) + Low;
}      


/******************************************************************************
               A P P L I C A T I O N - S P E C I F I C   C O D E
 ******************************************************************************/


#define NUM_LAYERS    3
#define N             30
#define M             1
INT                   Units[NUM_LAYERS] = {N, 10, M};

#define FIRST_YEAR    1700
#define NUM_YEARS     280

#define TRAIN_LWB     (N)
#define TRAIN_UPB     (179)
#define TRAIN_YEARS   (TRAIN_UPB - TRAIN_LWB + 1)
#define TEST_LWB      (180)
#define TEST_UPB      (259)
#define TEST_YEARS    (TEST_UPB - TEST_LWB + 1)
#define EVAL_LWB      (260)
#define EVAL_UPB      (NUM_YEARS - 1)
#define EVAL_YEARS    (EVAL_UPB - EVAL_LWB + 1)

REAL                  Sunspots_[NUM_YEARS];
REAL                  Sunspots [NUM_YEARS] = {

                        0.0262,  0.0575,  0.0837,  0.1203,  0.1883,  0.3033,  
                        0.1517,  0.1046,  0.0523,  0.0418,  0.0157,  0.0000,  
                        0.0000,  0.0105,  0.0575,  0.1412,  0.2458,  0.3295,  
                        0.3138,  0.2040,  0.1464,  0.1360,  0.1151,  0.0575,  
                        0.1098,  0.2092,  0.4079,  0.6381,  0.5387,  0.3818,  
                        0.2458,  0.1831,  0.0575,  0.0262,  0.0837,  0.1778,  
                        0.3661,  0.4236,  0.5805,  0.5282,  0.3818,  0.2092,  
                        0.1046,  0.0837,  0.0262,  0.0575,  0.1151,  0.2092,  
                        0.3138,  0.4231,  0.4362,  0.2495,  0.2500,  0.1606,  
                        0.0638,  0.0502,  0.0534,  0.1700,  0.2489,  0.2824,  
                        0.3290,  0.4493,  0.3201,  0.2359,  0.1904,  0.1093,  
                        0.0596,  0.1977,  0.3651,  0.5549,  0.5272,  0.4268,  
                        0.3478,  0.1820,  0.1600,  0.0366,  0.1036,  0.4838,  
                        0.8075,  0.6585,  0.4435,  0.3562,  0.2014,  0.1192,  
                        0.0534,  0.1260,  0.4336,  0.6904,  0.6846,  0.6177,  
                        0.4702,  0.3483,  0.3138,  0.2453,  0.2144,  0.1114,  
                        0.0837,  0.0335,  0.0214,  0.0356,  0.0758,  0.1778,  
                        0.2354,  0.2254,  0.2484,  0.2207,  0.1470,  0.0528,  
                        0.0424,  0.0131,  0.0000,  0.0073,  0.0262,  0.0638,  
                        0.0727,  0.1851,  0.2395,  0.2150,  0.1574,  0.1250,  
                        0.0816,  0.0345,  0.0209,  0.0094,  0.0445,  0.0868,  
                        0.1898,  0.2594,  0.3358,  0.3504,  0.3708,  0.2500,  
                        0.1438,  0.0445,  0.0690,  0.2976,  0.6354,  0.7233,  
                        0.5397,  0.4482,  0.3379,  0.1919,  0.1266,  0.0560,  
                        0.0785,  0.2097,  0.3216,  0.5152,  0.6522,  0.5036,  
                        0.3483,  0.3373,  0.2829,  0.2040,  0.1077,  0.0350,  
                        0.0225,  0.1187,  0.2866,  0.4906,  0.5010,  0.4038,  
                        0.3091,  0.2301,  0.2458,  0.1595,  0.0853,  0.0382,  
                        0.1966,  0.3870,  0.7270,  0.5816,  0.5314,  0.3462,  
                        0.2338,  0.0889,  0.0591,  0.0649,  0.0178,  0.0314,  
                        0.1689,  0.2840,  0.3122,  0.3332,  0.3321,  0.2730,  
                        0.1328,  0.0685,  0.0356,  0.0330,  0.0371,  0.1862,  
                        0.3818,  0.4451,  0.4079,  0.3347,  0.2186,  0.1370,  
                        0.1396,  0.0633,  0.0497,  0.0141,  0.0262,  0.1276,  
                        0.2197,  0.3321,  0.2814,  0.3243,  0.2537,  0.2296,  
                        0.0973,  0.0298,  0.0188,  0.0073,  0.0502,  0.2479,  
                        0.2986,  0.5434,  0.4215,  0.3326,  0.1966,  0.1365,  
                        0.0743,  0.0303,  0.0873,  0.2317,  0.3342,  0.3609,  
                        0.4069,  0.3394,  0.1867,  0.1109,  0.0581,  0.0298,  
                        0.0455,  0.1888,  0.4168,  0.5983,  0.5732,  0.4644,  
                        0.3546,  0.2484,  0.1600,  0.0853,  0.0502,  0.1736,  
                        0.4843,  0.7929,  0.7128,  0.7045,  0.4388,  0.3630,  
                        0.1647,  0.0727,  0.0230,  0.1987,  0.7411,  0.9947,  
                        0.9665,  0.8316,  0.5873,  0.2819,  0.1961,  0.1459,  
                        0.0534,  0.0790,  0.2458,  0.4906,  0.5539,  0.5518,  
                        0.5465,  0.3483,  0.3603,  0.1987,  0.1804,  0.0811,  
                        0.0659,  0.1428,  0.4838,  0.8127 

                      };

REAL                  Mean;
REAL                  TrainError;
REAL                  TrainErrorPredictingMean;
REAL                  TestError;
REAL                  TestErrorPredictingMean;

FILE*                 f;


void NormalizeSunspots()
{
  INT  Year;
  REAL Min, Max;
	
  Min = MAX_REAL;
  Max = MIN_REAL;
  for (Year=0; Year<NUM_YEARS; Year++) {
    Min = MIN(Min, Sunspots[Year]);
    Max = MAX(Max, Sunspots[Year]);
  }
  Mean = 0;
  for (Year=0; Year<NUM_YEARS; Year++) {
    Sunspots_[Year] = 
    Sunspots [Year] = ((Sunspots[Year]-Min) / (Max-Min)) * (HI-LO) + LO;
    Mean += Sunspots[Year] / NUM_YEARS;
  }
}


void InitializeApplication(NET* Net)
{
  INT  Year, i;
  REAL Out, Err;

  Net->Alpha = 0.5;
  Net->Eta   = 0.05;
  Net->Gain  = 1;

  NormalizeSunspots();
  TrainErrorPredictingMean = 0;
  for (Year=TRAIN_LWB; Year<=TRAIN_UPB; Year++) {
    for (i=0; i<M; i++) {
      Out = Sunspots[Year+i];
      Err = Mean - Out;
      TrainErrorPredictingMean += 0.5 * sqr(Err);
    }
  }
  TestErrorPredictingMean = 0;
  for (Year=TEST_LWB; Year<=TEST_UPB; Year++) {
    for (i=0; i<M; i++) {
      Out = Sunspots[Year+i];
      Err = Mean - Out;
      TestErrorPredictingMean += 0.5 * sqr(Err);
    }
  }
  f = fopen("BPN.txt", "w");
}


void FinalizeApplication(NET* Net)
{
  fclose(f);
}


/******************************************************************************
                          I N I T I A L I Z A T I O N
 ******************************************************************************/


void GenerateNetwork(NET* Net)
{
  INT l,i;

  Net->Layer = (LAYER**) calloc(NUM_LAYERS, sizeof(LAYER*));
   
  for (l=0; l<NUM_LAYERS; l++) {
    Net->Layer[l] = (LAYER*) malloc(sizeof(LAYER));
      
    Net->Layer[l]->Units      = Units[l];
    Net->Layer[l]->Output     = (REAL*)  calloc(Units[l]+1, sizeof(REAL));
    Net->Layer[l]->Error      = (REAL*)  calloc(Units[l]+1, sizeof(REAL));
    Net->Layer[l]->Weight     = (REAL**) calloc(Units[l]+1, sizeof(REAL*));
    Net->Layer[l]->WeightSave = (REAL**) calloc(Units[l]+1, sizeof(REAL*));
    Net->Layer[l]->dWeight    = (REAL**) calloc(Units[l]+1, sizeof(REAL*));
    Net->Layer[l]->Output[0]  = BIAS;
      
    if (l != 0) {
      for (i=1; i<=Units[l]; i++) {
        Net->Layer[l]->Weight[i]     = (REAL*) calloc(Units[l-1]+1, sizeof(REAL));
        Net->Layer[l]->WeightSave[i] = (REAL*) calloc(Units[l-1]+1, sizeof(REAL));
        Net->Layer[l]->dWeight[i]    = (REAL*) calloc(Units[l-1]+1, sizeof(REAL));
      }
    }
  }
  Net->InputLayer  = Net->Layer[0];
  Net->OutputLayer = Net->Layer[NUM_LAYERS - 1];
  Net->Alpha       = 0.9;
  Net->Eta         = 0.25;
  Net->Gain        = 1;
}


void RandomWeights(NET* Net)
{
  INT l,i,j;
   
  for (l=1; l<NUM_LAYERS; l++) {
    for (i=1; i<=Net->Layer[l]->Units; i++) {
      for (j=0; j<=Net->Layer[l-1]->Units; j++) {
        Net->Layer[l]->Weight[i][j] = RandomEqualREAL(-0.5, 0.5);
      }
    }
  }
}


void SetInput(NET* Net, REAL* Input)
{
  INT i;
   
  for (i=1; i<=Net->InputLayer->Units; i++) {
    Net->InputLayer->Output[i] = Input[i-1];
  }
}


void GetOutput(NET* Net, REAL* Output)
{
  INT i;
   
  for (i=1; i<=Net->OutputLayer->Units; i++) {
    Output[i-1] = Net->OutputLayer->Output[i];
  }
}


/******************************************************************************
            S U P P O R T   F O R   S T O P P E D   T R A I N I N G
 ******************************************************************************/


void SaveWeights(NET* Net)
{
  INT l,i,j;

  for (l=1; l<NUM_LAYERS; l++) {
    for (i=1; i<=Net->Layer[l]->Units; i++) {
      for (j=0; j<=Net->Layer[l-1]->Units; j++) {
        Net->Layer[l]->WeightSave[i][j] = Net->Layer[l]->Weight[i][j];
      }
    }
  }
}


void RestoreWeights(NET* Net)
{
  INT l,i,j;

  for (l=1; l<NUM_LAYERS; l++) {
    for (i=1; i<=Net->Layer[l]->Units; i++) {
      for (j=0; j<=Net->Layer[l-1]->Units; j++) {
        Net->Layer[l]->Weight[i][j] = Net->Layer[l]->WeightSave[i][j];
      }
    }
  }
}


/******************************************************************************
                     P R O P A G A T I N G   S I G N A L S
 ******************************************************************************/


// void PropagateLayer(NET* Net, LAYER* Lower, LAYER* Upper)
// {
//   INT  i,j;
//   REAL Sum;

//   for (i=1; i<=Upper->Units; i++) {
//     Sum = 0;
//     for (j=0; j<=Lower->Units; j++) {
//       Sum += Upper->Weight[i][j] * Lower->Output[j];
//     }
//     Upper->Output[i] = 1 / (1 + exp(-(Net->Gain) * Sum));
//   }
// }


// void PropagateNet(NET* Net)
// {
//   INT l;
   
//   for (l=0; l<NUM_LAYERS-1; l++) {
//     PropagateLayer(Net, Net->Layer[l], Net->Layer[l+1]);
//   }
// }


CUDA CODE FOR THIS LAYER

__device__ REAL sigmoid(REAL x, REAL gain) {
    return 1.0 / (1.0 + exp(-gain * x));
}



__global__ void propagateLayerKernel(REAL *lowerOutput, REAL *upperOutput, REAL *upperWeight,
                                     int lowerUnits, int upperUnits, REAL gain) {
    INT l = blockIdx.x * blockDim.x + threadIdx.x;

    if (l <= upperUnits) {
        REAL sum = 0;
        for(int j=0; j<= lowerUnits - 3; j+=4) {
            sum += upperWeight[l * (lowerUnits + 1) + j] * lowerOutput[j] +
            upperWeight[l * (lowerUnits + 1) + (j + 1)] * lowerOutput[j + 1] +
            upperWeight[l * (lowerUnits + 1) + (j + 2)] * lowerOutput[j + 2] +
            upperWeight[l * (lowerUnits + 1) + (j + 3)] * lowerOutput[j + 3];
        }

        //handle the rest
        for(int j= (lowerUnits / 4) * 4; j <= lowerUnits; j++) {
            sum += upperWeight[l * (lowerUnits + 1) + j] * lowerOutput[j];
        }

        upperOutput[l]  = sigmoid(sum, -gain);
        //upperOutput[l] = 1 / (1 + expf(-gain * sum)); //test
    }
}

void PropagateLayer(NET* Net, LAYER* Lower, LAYER* Upper) {
    REAL *d_lowerOutput, *d_upperOutput, *d_upperWeight;
    
    hipMalloc((void**)&d_lowerOutput, (Lower->Units + 1) * sizeof(REAL));
    hipMalloc((void**)&d_upperOutput, (Upper->Units + 1) * sizeof(REAL));
    hipMalloc((void**)&d_upperWeight, (Upper->Units + 1) * (Lower->Units + 1) * sizeof(REAL));
   
    hipMemcpy(d_lowerOutput, Lower->Output, (Lower->Units + 1) * sizeof(REAL), hipMemcpyHostToDevice);
    hipMemcpy(d_upperWeight, Upper->Wesight, (Upper->Units + 1) * (Lower->Units + 1) * sizeof(REAL), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockDim(256);
    dim3 gridDim((Upper->Units + blockDim.x) / blockDim.x);

    propagateLayerKernel<<<gridDim, blockDim>>>(d_lowerOutput, d_upperOutput, d_upperWeight,
                                                Lower->Units, Upper->Units, Net->Gain);
    hipDeviceSynchronize();
    // Copy results back
    hipMemcpy(Upper->Output, d_upperOutput, (Upper->Units + 1) * sizeof(REAL), hipMemcpyDeviceToHost);

    // Free memory on GPU
    hipFree(d_lowerOutput);
    hipFree(d_upperOutput);
    hipFree(d_upperWeight);
}

void PropagateNet(NET* Net)
{
  INT l;
   
  for (l=0; l<NUM_LAYERS-1; l++) {
    PropagateLayer(Net, Net->Layer[l], Net->Layer[l+1]);
  }
}



/******************************************************************************
                  B A C K P R O P A G A T I N G   E R R O R S
 ******************************************************************************/


void ComputeOutputError(NET* Net, REAL* Target)
{
  INT  i;
  REAL Out, Err;
   
  Net->Error = 0;
  for (i=1; i<=Net->OutputLayer->Units; i++) {
    Out = Net->OutputLayer->Output[i];
    Err = Target[i-1]-Out;
    Net->OutputLayer->Error[i] = Net->Gain * Out * (1-Out) * Err;
    Net->Error += 0.5 * sqr(Err);
  }
}


void BackpropagateLayer(NET* Net, LAYER* Upper, LAYER* Lower)
{
  INT  i,j;
  REAL Out, Err;
   
  for (i=1; i<=Lower->Units; i++) {
    Out = Lower->Output[i];
    Err = 0;
    for (j=1; j<=Upper->Units; j++) {
      Err += Upper->Weight[j][i] * Upper->Error[j];
    }
    Lower->Error[i] = Net->Gain * Out * (1-Out) * Err;
  }
}


void BackpropagateNet(NET* Net)
{
  INT l;
   
  for (l=NUM_LAYERS-1; l>1; l--) {
    BackpropagateLayer(Net, Net->Layer[l], Net->Layer[l-1]);
  }
}


void AdjustWeights(NET* Net)
{
  INT  l,i,j;
  REAL Out, Err, dWeight;
   
  for (l=1; l<NUM_LAYERS; l++) {
    for (i=1; i<=Net->Layer[l]->Units; i++) {
      for (j=0; j<=Net->Layer[l-1]->Units; j++) {
        Out = Net->Layer[l-1]->Output[j];
        Err = Net->Layer[l]->Error[i];
        dWeight = Net->Layer[l]->dWeight[i][j];
        Net->Layer[l]->Weight[i][j] += Net->Eta * Err * Out + Net->Alpha * dWeight;
        Net->Layer[l]->dWeight[i][j] = Net->Eta * Err * Out;
      }
    }
  }
}


/******************************************************************************
                      S I M U L A T I N G   T H E   N E T
 ******************************************************************************/


void SimulateNet(NET* Net, REAL* Input, REAL* Output, REAL* Target, BOOL Training)
{
  SetInput(Net, Input);
  PropagateNet(Net);
  GetOutput(Net, Output);
   
  ComputeOutputError(Net, Target);
  if (Training) {
    BackpropagateNet(Net);
    AdjustWeights(Net);
  }
}


void TrainNet(NET* Net, INT Epochs)
{
  INT  Year, n;
  REAL Output[M];

  for (n=0; n<Epochs*TRAIN_YEARS; n++) {
    Year = RandomEqualINT(TRAIN_LWB, TRAIN_UPB);
    SimulateNet(Net, &(Sunspots[Year-N]), Output, &(Sunspots[Year]), TRUE);
  }
}


void TestNet(NET* Net)
{
  INT  Year;
  REAL Output[M];

  TrainError = 0;
  for (Year=TRAIN_LWB; Year<=TRAIN_UPB; Year++) {
    SimulateNet(Net, &(Sunspots[Year-N]), Output, &(Sunspots[Year]), FALSE);
    TrainError += Net->Error;
  }
  TestError = 0;
  for (Year=TEST_LWB; Year<=TEST_UPB; Year++) {
    SimulateNet(Net, &(Sunspots[Year-N]), Output, &(Sunspots[Year]), FALSE);
    TestError += Net->Error;
  }
  fprintf(f, "\nNMSE is %0.3f on Training Set and %0.3f on Test Set",
             TrainError / TrainErrorPredictingMean,
             TestError / TestErrorPredictingMean);
}


void EvaluateNet(NET* Net)
{
  INT  Year;
  REAL Output [M];
  REAL Output_[M];

  fprintf(f, "\n\n\n");
  fprintf(f, "Year    Sunspots    Open-Loop Prediction    Closed-Loop Prediction\n");
  fprintf(f, "\n");
  for (Year=EVAL_LWB; Year<=EVAL_UPB; Year++) {
    SimulateNet(Net, &(Sunspots [Year-N]), Output,  &(Sunspots [Year]), FALSE);
    SimulateNet(Net, &(Sunspots_[Year-N]), Output_, &(Sunspots_[Year]), FALSE);
    Sunspots_[Year] = Output_[0];
    fprintf(f, "%d       %0.3f                   %0.3f                     %0.3f\n",
               FIRST_YEAR + Year,
               Sunspots[Year],
               Output [0],
               Output_[0]);
  }
}


/******************************************************************************
                                    M A I N
 ******************************************************************************/


int main()
{
  NET  Net;
  BOOL Stop;
  REAL MinTestError;

  InitializeRandoms();
  GenerateNetwork(&Net);
  RandomWeights(&Net);
  InitializeApplication(&Net);

  Stop = FALSE;
  MinTestError = MAX_REAL;
  do {
    TrainNet(&Net, 10);
    TestNet(&Net);
    if (TestError < MinTestError) {
      fprintf(f, " - saving Weights ...");
      MinTestError = TestError;
      SaveWeights(&Net);
    }
    else if (TestError > 1.2 * MinTestError) {
      fprintf(f, " - stopping Training and restoring Weights ...");
      Stop = TRUE;
      RestoreWeights(&Net);
    }
  } while (NOT Stop);

  TestNet(&Net);
  EvaluateNet(&Net);
   
  FinalizeApplication(&Net);
  return 0;
}

