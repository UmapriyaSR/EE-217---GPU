#include "hip/hip_runtime.h"
#include "declaration.h"
#define BLOCK_SIZE 256
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include<iostream>
#include <math.h>

/******************************************************************************
        R A N D O M S   D R A W N   F R O M   D I S T R I B U T I O N S
 ******************************************************************************/
__device__ double atomicAddDouble(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);

    return __longlong_as_double(old);
}





__global__ void initRandomsKernel(hiprandState *state, unsigned long seed) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, id, 0, &state[id]);
}

__global__ void generateRandomIntsKernel(hiprandState *state, int *output, int low, int high, int num) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < num) {
        output[id] = low + (int)(hiprand_uniform(&state[id]) * (high - low + 1));
    }
}

__global__ void generateRandomRealsKernel(hiprandState *state, REAL *output, REAL low, REAL high, int num) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < num) {
        output[id] = low + hiprand_uniform(&state[id]) * (high - low);
    }
}



/*__device__ float RandomEqualREAL_GPU(float min, float max)
{
    hiprandState state;
    hiprand_init(1234, threadIdx.x, 0, &state);  // Adjust the seed (1234 in this example)
    
    // Generate a random value between min and max
    float randomValue = hiprand_uniform(&state) * (max - min) + min;
    
    return randomValue;
} //temp


/******************************************************************************
               A P P L I C A T I O N - S P E C I F I C   C O D E
 ******************************************************************************/

__global__ void findMaxKernel(REAL *sunspots, REAL *max,int numyears) {
    extern __shared__ REAL sdata[];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load shared memory
    sdata[tid] = (i < numyears) ? sunspots[i] : MIN_REAL;
    __syncthreads();

    // Perform reduction in shared memory
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = (sdata[tid] > sdata[tid + s]) ? sdata[tid] : sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) max[blockIdx.x] = sdata[0];
}

__global__ void findMinKernel(REAL *sunspots, REAL *min,int numyears) {
    extern __shared__ REAL sdata[];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load shared memory
    sdata[tid] = (i < numyears) ? sunspots[i] : MAX_REAL;
    __syncthreads();

    // Perform reduction in shared memory
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = (sdata[tid] < sdata[tid + s]) ? sdata[tid] : sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) min[blockIdx.x] = sdata[0];
}

__global__ void normalizeSunspotsKernel(REAL *sunspots, REAL min, REAL max, REAL *mean, int numYears, REAL lo, REAL hi) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < numYears) {
        sunspots[index] = ((sunspots[index] - min) / (max - min)) * (hi - lo) + lo;
        atomicAdd(mean, sunspots[index] / numYears); // Use atomicAdd for concurrent addition
    }
}
__global__ void computeErrorKernel(REAL *sunspots, REAL mean, REAL *error, int startYear, int endYear,int m) {
   int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index + startYear <= endYear) {
        for (int i = 0; i < m; i++) {
            REAL out = sunspots[index + startYear + i];
            REAL err = mean - out;
            atomicAdd(error, 0.5 * err * err); // Atomic add to avoid race conditions
        }
    }
}
/******************************************************************************
                          I N I T I A L I Z A T I O N
 ******************************************************************************/
__global__ void initializeRandomWeights(REAL** weight, int numUnitsPrevLayer, int numUnitsCurrLayer, unsigned long seed) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < numUnitsCurrLayer && j < (numUnitsPrevLayer + 1)) {
        int idx = i * (numUnitsPrevLayer + 1) + j;

        hiprandState state;
        hiprand_init(seed, idx, 0, &state);

        weight[i][j] = hiprand_uniform(&state) - 0.5; // Generates values between -0.5 and 0.5
    }
}
/******************************************************i************************
            S U P P O R T   F O R   S T O P P E D   T R A I N I N G
 ******************************************************************************/

__global__ void saveWeightsKernel(REAL** weight, REAL** weightSave, int numUnitsPrevLayer, int numUnitsCurrLayer) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < numUnitsCurrLayer && j < (numUnitsPrevLayer + 1)) {
        //int idx = i * (numUnitsPrevLayer + 1) + j;
        weightSave[i][j] = weight[i][j];
    }
}


__global__ void restoreWeightsKernel(REAL** weight, REAL** weightSave, int numRows, int numCols) {
    int i = blockIdx.y * blockDim.y + threadIdx.y; // Row index
    int j = blockIdx.x * blockDim.x + threadIdx.x; // Column index

    if (i < numRows && j < numCols) {
        weight[i][j] = weightSave[i][j];
    }
}
/******************************************************************************
                     P R O P A G A T I N G   S I G N A L S
 ******************************************************************************/

/*
__global__ void backpropagateLayerKernel(REAL* lowerOutput, REAL* upperWeight, REAL* upperError, REAL* lowerError, int lowerUnits, int upperUnits, REAL gain) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < lowerUnits) {
        REAL out = lowerOutput[i + 1]; // Assuming output array starts at index 1
        REAL err = 0;
        for (int j = 1; j <= upperUnits; j++) {
            err += upperWeight[j * (lowerUnits + 1) + i] * upperError[j];
        }
        lowerError[i + 1] = gain * out * (1 - out) * err;
    }
}*/

__device__ REAL sigmoid(REAL x, REAL gain) {
    return 1.0 / (1.0 + exp(-gain * x));
}

__global__ void propagateNetKernel(NET* net) {
    INT l = blockIdx.x * blockDim.x + threadIdx.x;

    if (l < NUM_LAYERS - 1) {
        LAYER* lower = net->Layer[l];
        LAYER* upper = net->Layer[l + 1];

        INT i = blockIdx.x * blockDim.x + threadIdx.x + 1;

        if (i <= upper->Units) {
            REAL sum = 0;
            for (INT j = 0; j <= lower->Units; j+= 2) {
                //loop unrolling optimization
                sum += upper->Weight[i][j] * lower->Output[j];
                sum += upper->Weight[i][j+1] * lower->Output[j+1];
            }
            upper->Output[i] = sigmoid(sum, net->Gain);
        }
    }
}

/******************************************************************************
                  B A C K P R O P A G A T I N G   E R R O R S
 ******************************************************************************/

/*
__global__ void ComputeOutputErrorKernel(REAL *output, REAL *target, REAL *error, int units, REAL gain, REAL *netError) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < units) {
        REAL out = output[i];
        REAL err = target[i] - out;  // Assuming zero-based indexing for target
        error[i] = gain * out * (1 - out) * err;
        atomicAdd(netError, 0.5 * err * err); // Accumulate the squared error
    }
}

__global__ void BackpropagateLayerKernel(REAL *lowerOutput, REAL *lowerError, REAL **upperWeight, REAL *upperError, int lowerUnits, int upperUnits, REAL gain) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < lowerUnits) {
        REAL out = lowerOutput[i];
        REAL err = 0;
        for (int j = 0; j < upperUnits; j++) {
            err += upperWeight[j][i] * upperError[j];
        }
        lowerError[i] = gain * out * (1 - out) * err;
    }
}

__global__ void adjustWeightsKernel(REAL* lowerOutput, REAL* upperError, REAL* weight, REAL* dWeight, int lowerUnits, int upperUnits, REAL eta, REAL alpha) {
    int i = blockIdx.y * blockDim.y + threadIdx.y; // For upper units
    int j = blockIdx.x * blockDim.x + threadIdx.x; // For lower units

    if (i <= upperUnits && j <= lowerUnits) {
        REAL out = lowerOutput[j];
        REAL err = upperError[i];
        REAL dw = dWeight[i * (lowerUnits + 1) + j];
        weight[i * (lowerUnits + 1) + j] += eta * err * out + alpha * dw;
        dWeight[i * (lowerUnits + 1) + j] = eta * err * out;
    }
}


/******************************************************************************
                      S I M U L A T I N G   T H E   N E T
 ******************************************************************************/
